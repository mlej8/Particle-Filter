#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "robot.h"

using namespace std;

__global__ void simulate_robot_motion() {}

__global__ void importance_weight_computation() {}

__global__ void resampling() {}

int main() {
  // physical robot (ground truth)
  Robot my_robot;

  // number of particles
  int N = 1000;

  // number of iterations of particle filter
  int T = 10;

  // initialize N random particles (robots)
  // list of particles  (guesses as to where the robot might be - each particle
  // is a vector representing the state of the robot (x,y,theta)  theta is the
  // angle relative to the x-axis)
  std::vector<Robot> particles;
  for (int i = 0; i < N; i++) {
    Robot r;
    r.set_noise(0.05, 0.05, 5.0);
    particles.push_back(r);
  }

  for (int j = 0; j < T; j++) {
    // TODO here we are always turning by 0.1 and moving for 5 meters, randomly
    // generate the movement of the True robot
    my_robot = my_robot.move(0.1, 5.0);
    vector<double> Z = my_robot.sense();

    // TODO take each of the particles and simulate robot motion
    std::vector<Robot> p2;
    for (int k = 0; k < N; k++) {
      p2.push_back(particles[k].move(0.1, 5.0));
    }
    particles = p2;

    // TODO weigth computation
    std::vector<double> w;
    for (int k = 0; k < N; k++) {
      w.push_back(particles[k].measurement_prob(Z));
    }

    std::vector<Robot> p3;

    // resampling
    int index = (int)(uniform_distribution(generator) * N);
    double beta = 0.0;
    double max_w = w[0];
    for (int l = 0; l < w.size(); l++) {
      if (w[l] > max_w) {
        max_w = w[l];
      }
    }

    for (int m = 0; m < N; m++) {
      beta += uniform_distribution(generator) * 2.0 * max_w;
      while (beta > w[index]) {
        beta -= w[index];
        index = (index + 1) % N;
      }
      p3.push_back(particles[index]);
    }
    particles = p3;

    cout << eval(my_robot, particles) << endl;
  }
  return 0;
}