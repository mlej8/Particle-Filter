#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "robot.cuh"

using namespace std;

// TODO fix bug where if block_size > 512 code fails

__global__ void weight_normalization(double *weights, double weight_sum, const int N){
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) {
    weights[index] /= weight_sum;
  }
}

/**
 * Simulates robot motion for each particle, performs measurement update and
 * assigns a weight to each particle.
 */
__global__ void particle_filter(Robot *particles, double *weights,
                                const double theta, const double distance,
                                const int N, const double *Z,
                                const int num_landmarks,
                                const double *landmarks_gpu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) {
    particles[index].move(theta, distance);

    // likelihood of the real observation based on the current particle's state
    double prob = 1.0;
    for (int i = 0; i < num_landmarks; i++) {
      double dist =
          sqrt((particles[index].get_x() - landmarks_gpu[i * 2 + 0]) *
                   (particles[index].get_x() - landmarks_gpu[i * 2 + 0]) +
               (particles[index].get_y() - landmarks_gpu[i * 2 + 1]) *
                   (particles[index].get_y() - landmarks_gpu[i * 2 + 1]));
      prob *= Gaussian(dist, particles[index].get_sense_noise(), Z[i]);
    }
    weights[index] = prob;
  }
}
int main(int argc, char *argv[]) {
  // physical robot (ground truth)
  auto start = chrono::high_resolution_clock::now();

  if (argc != 4) {
    cout << "Usage: ./particle_filter_gpu <number of particles> <number of "
            "iterations of particle filtering> <number of threads per block>"
         << endl;
    exit(1);
  }

  // physical robot (ground truth)
  Robot my_robot;

  // number of particles (TODO: set default to 1000)
  int N = atoi(argv[1]);

  // number of particle filter iterations (TODO: set default to 10)
  int T = atoi(argv[2]);

  int block_size = atoi(argv[3]);
  size_t num_block = (N + block_size - 1) / block_size;
  size_t particles_size = N * sizeof(Robot);
  int num_landmarks = sizeof(landmarks) / sizeof(landmarks[0]);
  int landmark_dim = sizeof(landmarks[0]) / sizeof(double);
  size_t landmark_size = sizeof(double) * num_landmarks * landmark_dim;

  // initialize N random particles (robots)
  // list of particles (guesses as to where the robot might be - each particle
  // is a vector representing the state of the robot (x,y,theta) theta is the
  // angle relative to the x-axis)
  vector<Robot> particles(N);
  Robot *particles_gpu;
  thrust::device_vector<double> weights_gpu(N);
  thrust::host_vector<double> weights(N);
  double *landmarks_gpu;
  hipMalloc(&landmarks_gpu, landmark_size);
  hipMalloc(&particles_gpu, particles_size);
  
  hipMemcpy(landmarks_gpu, landmarks, landmark_size, hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu, particles.data(), particles_size,
             hipMemcpyHostToDevice);

  for (int j = 0; j < T; j++) {
    double theta = uniform_distribution_sample() * M_PI / 2;
    double distance = (uniform_distribution_sample() * 9.0) + 1;
    my_robot.move(theta, distance);

    // detect distance of the real robot to the landmarks in our world (returns
    // list of distance to each obstacle)
    vector<double> Z = my_robot.sense();
    thrust::device_vector<double> Z_gpu(Z);

    particle_filter<<<num_block, block_size>>>(
        particles_gpu, thrust::raw_pointer_cast(weights_gpu.data()), theta,
        distance, N, thrust::raw_pointer_cast(Z_gpu.data()), Z.size(),
        landmarks_gpu);
    hipDeviceSynchronize();

    // weights summation via parallelization
    double weights_sum = thrust::reduce(weights_gpu.begin(), weights_gpu.end()); // , thrust::plus<double>()

    weight_normalization<<<num_block, block_size>>>(thrust::raw_pointer_cast(weights_gpu.data()), weights_sum, N);
    hipDeviceSynchronize();

    // copying normalized particles' weights from GPU to CPU
    thrust::copy(weights_gpu.begin(), weights_gpu.end(), weights.begin());

    // copying updated particles to CPU
    hipMemcpy(particles.data(), particles_gpu, particles_size,
               hipMemcpyDeviceToHost);

    // compute cumulative distribution function (CDF)
    thrust::device_vector<double> cdf_gpu(weights_gpu.size());
    thrust::inclusive_scan(weights_gpu.begin(), weights_gpu.end(), cdf_gpu.begin());
    thrust::host_vector<double> cdf(cdf_gpu);

    int k = 0;
    auto u = [&N](int n) {
      return (((n - 1) + uniform_distribution_sample()) / N);
    };

    // systematic resampling of new particles
    vector<Robot> new_particles;
    for (int i = 1; i <= N; i++) {
      while (cdf[k] < u(i)) {
        k += 1;
      }
      new_particles.push_back(particles[k]);
    }
    particles = new_particles;

    // copy new particles to GPU for next iteration
    hipMemcpy(particles_gpu, particles.data(), particles_size,
               hipMemcpyHostToDevice);

    cout << eval(my_robot, particles) << endl;
  }

  hipFree(particles_gpu);
  hipFree(landmarks_gpu);
  auto finish = chrono::high_resolution_clock::now();
  std::cout
      << block_size << "," << N << ","
      << chrono::duration_cast<chrono::nanoseconds>(finish - start).count()
      << "\n";

  return 0;
}