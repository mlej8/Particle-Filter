#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>

#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "robot.cuh"

using namespace std;

/**
 * Simulate robot motion for each particle and perform importance weight
 * computation
 */
__global__ void particle_filter(Robot *particles, double *weights,
                                const double theta, const double distance,
                                const int N, const double *Z_gpu,
                                const int num_landmarks,
                                const double *landmarks) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) {
    printf("???");
    particles[index].move(theta, distance);
    double prob = 1.0;
    for (int i = 0; i < num_landmarks; i++) {
      double dist = sqrt((particles[index].get_x() - landmarks[i * 2 + 0]) *
          (particles[index].get_x() - landmarks[i * 2 + 0]) +
          (particles[index].get_y() - landmarks[i * 2 + 1]) *
              (particles[index].get_y() - landmarks[i * 2 + 1]));
      prob *= Gaussian(dist, particles[index].get_sense_noise(), Z_gpu[i]);
    }
    weights[index] = prob;
  }
}
int main(int argc, char *argv[]) {
  // physical robot (ground truth)
  Robot my_robot;

  if (argc != 4) {
    cout << "Usage: ./particle_filter_gpu <number of particles> <number of "
            "iterations of particle filtering> <number of threads per block>"
         << endl;
    exit(1);
  }

  // number of particles (TODO: set default to 1000)
  int N = atoi(argv[1]);

  // number of iterations of particle filter (TODO: set default to 10)
  int T = atoi(argv[2]);

  int block_size = atoi(argv[3]);
  size_t num_block = (N + block_size - 1) / block_size;
  size_t particles_size = N * sizeof(Robot);
  int num_landmarks = sizeof(landmarks) / sizeof(landmarks[0]);
  int landmark_dim = sizeof(landmarks[0]) / sizeof(double);
  size_t landmark_size = sizeof(double) * num_landmarks * landmark_dim;

  // initialize N random particles (robots)
  // list of particles  (guesses as to where the robot might be - each particle
  // is a vector representing the state of the robot (x,y,theta)  theta is the
  // angle relative to the x-axis)
  vector<Robot> particles(N);
  Robot *particles_gpu;
  thrust::device_vector<double> weights_gpu(N);
  thrust::host_vector<double> weights(N);
  double *landmarks_gpu;
  hipMalloc(&landmarks_gpu, landmark_size);
  hipMemcpy(landmarks_gpu, landmarks, landmark_size, hipMemcpyHostToDevice);

  // copy those particles on the GPU
  hipMalloc(&particles_gpu, particles_size);
  // hipMallocManaged(&weights, N * sizeof(double));
  hipMemcpy(particles_gpu, particles.data(), particles_size,
             hipMemcpyHostToDevice);

  for (int j = 0; j < T; j++) {
    double theta = uniform_distribution_sample() * M_PI / 2;
    double distance = (uniform_distribution_sample() * 9.0) + 1;
    my_robot.move(theta, distance);
    vector<double> Z = my_robot.sense();
    thrust::device_vector<double> Z_gpu(Z);

    particle_filter<<<num_block, block_size>>>(
        particles_gpu, thrust::raw_pointer_cast(weights_gpu.data()), theta,
        distance, N, thrust::raw_pointer_cast(Z_gpu.data()), Z.size(),
        landmarks_gpu);
    hipDeviceSynchronize();

    thrust::copy(weights_gpu.begin(), weights_gpu.end(), weights.begin());
    double max_w = *(thrust::max_element(weights.begin(), weights.end()));

    hipMemcpy(particles.data(), particles_gpu, particles_size,
               hipMemcpyDeviceToHost);

    // resampling
    vector<Robot> new_particles;
    int index = (int) (uniform_distribution_sample() * N);
    double beta = 0.0;
    for (int m = 0; m < N; m++) {
      beta += uniform_distribution_sample() * 2.0 * max_w;
      while (beta > weights[index]) {
        beta -= weights[index];
        index = (index + 1) % N;
      }
      new_particles.push_back(particles[index]);
    }
    particles = new_particles;
    hipMemcpy(particles_gpu, particles.data(), particles_size,
               hipMemcpyHostToDevice);

    cout << eval(my_robot, particles) << endl;
  }

  hipFree(particles_gpu);
  return 0;
}