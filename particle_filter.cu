#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "robot.cuh"

using namespace std;

// TODO fix bug where if block_size > 512 code fails

__global__ void weight_normalization(double *weights, double weight_sum, const int N){
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) {
    weights[index] /= weight_sum;
  }
}

/**
 * Simulates robot motion for each particle, performs measurement update and
 * assigns a weight to each particle.
 */
__global__ void particle_filter(Robot *particles, double *weights,
                                const double theta, const double distance,
                                const int N, const double *Z,
                                const int num_landmarks,
                                const double *landmarks_gpu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) {
    particles[index].move(theta, distance);

    // likelihood of the real observation based on the current particle's state
    double prob = 1.0;
    for (int i = 0; i < num_landmarks; i++) {
      double dist =
          sqrt((particles[index].get_x() - landmarks_gpu[i * 2 + 0]) *
                   (particles[index].get_x() - landmarks_gpu[i * 2 + 0]) +
               (particles[index].get_y() - landmarks_gpu[i * 2 + 1]) *
                   (particles[index].get_y() - landmarks_gpu[i * 2 + 1]));
      prob *= Gaussian(dist, particles[index].get_sense_noise(), Z[i]);
    }
    weights[index] = prob;
  }
}
int main(int argc, char *argv[]) {
  // physical robot (ground truth)
  auto start = chrono::high_resolution_clock::now();

  if (argc == 2 || argc > 4) {
    cout << "Usage: ./particle_filter_gpu <optional:number_iteration "
            "(default:1000)> <optional:number_particles (default:10)>"
            "<number of threads per block (default:64)>"
         << endl;
    exit(1);
  }

  // physical robot (ground truth)
  Robot my_robot;
  my_robot.set_x(world_size/2);
  my_robot.set_y(world_size/2);

  // number of particles
  int N = 1000;

  // number of PF iterations
  int T = 10;

  int block_size = 64;

  if (argc == 4) {
    N = atoi(argv[1]);
    T = atoi(argv[2]);
    block_size = atoi(argv[3]);
  }

  size_t num_block = (N + block_size - 1) / block_size;
  size_t particles_size = N * sizeof(Robot);
  int num_landmarks = sizeof(landmarks) / sizeof(landmarks[0]);
  int landmark_dim = sizeof(landmarks[0]) / sizeof(double);
  size_t landmark_size = sizeof(double) * num_landmarks * landmark_dim;

  // initialize N random particles (robots)
  // list of particles (guesses as to where the robot might be - each particle
  // is a vector representing the state of the robot (x,y,theta) theta is the
  // angle relative to the x-axis)
  vector<Robot> particles(N);
  Robot *particles_gpu;
  thrust::device_vector<double> weights_gpu(N);
  thrust::host_vector<double> weights(N);
  double *landmarks_gpu;
  hipMalloc(&landmarks_gpu, landmark_size);
  hipMalloc(&particles_gpu, particles_size);
  
  hipMemcpy(landmarks_gpu, landmarks, landmark_size, hipMemcpyHostToDevice);

  for (int j = 0; j < T; j++) {
    double theta = uniform_distribution_sample() * M_PI / 2;
    double distance = (uniform_distribution_sample() * 9.0) + 1;
    my_robot.move(theta, distance);

    // detect distance of the real robot to the landmarks in our world (returns
    // list of distance to each obstacle)
    vector<double> Z = my_robot.sense();
    thrust::device_vector<double> Z_gpu(Z);

    // copy data to gpu
    hipMemcpy(particles_gpu, particles.data(), particles_size,
               hipMemcpyHostToDevice);

    particle_filter<<<num_block, block_size>>>(
        particles_gpu, thrust::raw_pointer_cast(weights_gpu.data()), theta,
        distance, N, thrust::raw_pointer_cast(Z_gpu.data()), Z.size(),
        landmarks_gpu);
    hipDeviceSynchronize();

    // weights summation via parallelization
    double weights_sum = thrust::reduce(weights_gpu.begin(), weights_gpu.end()); // , thrust::plus<double>()

    weight_normalization<<<num_block, block_size>>>(thrust::raw_pointer_cast(weights_gpu.data()), weights_sum, N);
    hipDeviceSynchronize();

    // copying normalized particles' weights from GPU to CPU
    thrust::copy(weights_gpu.begin(), weights_gpu.end(), weights.begin());

    // copying updated particles to CPU
    hipMemcpy(particles.data(), particles_gpu, particles_size,
               hipMemcpyDeviceToHost);

    // compute cumulative distribution function (CDF)
    thrust::device_vector<double> cdf_gpu(weights_gpu.size());
    thrust::inclusive_scan(weights_gpu.begin(), weights_gpu.end(), cdf_gpu.begin());
    thrust::host_vector<double> cdf(cdf_gpu);

    int k = 0;
    auto u = [&N](int n) {
      return (((n - 1) + uniform_distribution_sample()) / N);
    };

    // systematic resampling of new particles
    vector<Robot> new_particles;
    for (int i = 1; i <= N; i++) {
      while (cdf[k] < u(i)) {
        k += 1;
      }
      new_particles.push_back(particles[k]);
    }
    particles = new_particles;
    
    cout << eval(my_robot, particles, j) << endl;
  }

  hipFree(particles_gpu);
  hipFree(landmarks_gpu);
  auto finish = chrono::high_resolution_clock::now();
  std::cout
      << block_size << "," << N << ","
      << chrono::duration_cast<chrono::nanoseconds>(finish - start).count()
      << "\n";

  return 0;
}