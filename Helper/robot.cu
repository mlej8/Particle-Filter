#include "hip/hip_runtime.h"
#include <random>
#include <stdexcept>

#include "robot.cuh"

using namespace std;

static default_random_engine &get_engine() {
  // Initialized upon first call to the function.
  static default_random_engine engine;
  return engine;
}

static double uniform_distribution_sample() {
  // Initialized upon first call to the function.
  static uniform_real_distribution<double> uniform_distribution(0.0, 1.0);
  return uniform_distribution(get_engine());
}

Robot::Robot()
    : x(uniform_distribution_sample() * world_size),
      y(uniform_distribution_sample() * world_size),
      orientation(uniform_distribution_sample() * 2.0 * M_PI),
      forward_noise(0.05),
      turn_noise(0.05),
      sense_noise(5.0) {}

__host__ 
void Robot::set(double new_x, double new_y, double new_orientation) {
  if (new_x < 0 || new_x >= world_size) {
    throw invalid_argument("X coordinate out of bound");
  }
  if (new_y < 0 || new_y >= world_size) {
    throw invalid_argument("Y coordinate out of bound");
  }
  if (new_orientation < 0 || new_orientation >= 2 * M_PI) {
    throw invalid_argument("Orientation must be in [0..2pi]");
  }
  set_x(new_x);
  set_y(new_y);
  set_orientation(new_orientation);
}

__host__ 
void Robot::set_noise(double f_noise, double t_noise, double s_noise) {
  set_forward_noise(f_noise);
  set_turn_noise(t_noise);
  set_sense_noise(s_noise);
}

__host__ 
vector<double> Robot::sense() {
  vector<double> Z;
  for (int i = 0; i < (sizeof landmarks / sizeof landmarks[0]); i++) {
    double dist =
        sqrt((get_x() - landmarks[i][0]) * (get_x() - landmarks[i][0]) +
             (get_y() - landmarks[i][1]) * (get_y() - landmarks[i][1]));
    normal_distribution<double> distribution(0.0, get_forward_noise());
    dist += distribution(get_engine());
    Z.push_back(dist);
  }
  return Z;
}

__host__ __device__ 
void Robot::move(double turn, double forward) {
  if (forward < 0) {
    // throw invalid_argument("Robot cant move backwards");
    forward = 0;
  }
  normal_distribution<double> distribution1(0.0, get_turn_noise()); // TODO use cuRAND instaed
  normal_distribution<double> distribution2(0.0, get_forward_noise());
  orientation = orientation + turn + distribution1(get_engine());
  orientation = fmod(orientation, 2 * M_PI);
  if (orientation < 0) {
    orientation = 0.0;
  }
  double dist = forward + distribution2(get_engine());
  x = x + cos(orientation) * dist;
  y = y + sin(orientation) * dist;
  x = fmod(x, world_size);
  y = fmod(y, world_size);
  if (x < 0) {
    x = 0.0;
  }
  if (y < 0) {
    y = 0.0;
  }
}

__host__ 
double Robot::measurement_prob(vector<double> measurement) {
  double prob = 1.0;
  for (int i = 0; i < (sizeof landmarks / sizeof landmarks[0]); i++) {
    double dist =
        sqrt((get_x() - landmarks[i][0]) * (get_x() - landmarks[i][0]) +
             (get_y() - landmarks[i][1]) * (get_y() - landmarks[i][1]));
    prob *= Gaussian(dist, get_sense_noise(), measurement[i]);
  }
  return prob;
}

__host__ __device__ double Gaussian(double mu, double sigma, double x) {
  return exp(-((mu - x) * (mu - x)) / (sigma * sigma) / 2.0) /
         sqrt(2.0 * M_PI * (sigma * sigma));
}

__host__ double eval(Robot r, vector<Robot> p) {
  double sum = 0.0;
  for (int i = 0; i < p.size(); i++) {
    double dx = (p[i].get_x() - r.get_x() + fmod(world_size / 2.0, world_size) -
                 (world_size / 2.0));
    double dy = (p[i].get_y() - r.get_y() + fmod(world_size / 2.0, world_size) -
                 (world_size / 2.0));
    double err = sqrt(dx * dx + dy * dy);
    sum += err;
  }
  return sum / (double)p.size();
}

__host__ __device__ double Robot::get_x() const { return x; }
__host__ __device__ void Robot::set_x(double x) { Robot::x = x; }
__host__ __device__ double Robot::get_y() const { return y; }
__host__ __device__ void Robot::set_y(double y) { Robot::y = y; }
__host__ __device__ double Robot::get_orientation() const {
  return orientation;
}
__host__ __device__ void Robot::set_orientation(double orientation) {
  Robot::orientation = orientation;
}
__host__ __device__ double Robot::get_forward_noise() const {
  return forward_noise;
}
__host__ __device__ void Robot::set_forward_noise(double forward_noise) {
  Robot::forward_noise = forward_noise;
}
__host__ __device__ double Robot::get_turn_noise() const { return turn_noise; }
__host__ __device__ void Robot::set_turn_noise(double turn_noise) {
  Robot::turn_noise = turn_noise;
}
__host__ __device__ double Robot::get_sense_noise() const {
  return sense_noise;
}

__host__ __device__ void Robot::set_sense_noise(double sense_noise) {
  Robot::sense_noise = sense_noise;
}
